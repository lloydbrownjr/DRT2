#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <string>
#include <time.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>
#include <vector>
#include "../common/vec3.h"
#include "../common/ray.h"
#include "sphere.h"
#include "hitable_list.h"
#include "camera.h"
#include "material.h"
#include "cuda_errors.h"
#include "../common/options.h"
#include "hip/hip_runtime.h"
// #include "nccl.h"
#include "/opt/packages/mvapich2/intel/2.3.5-intel20.4/include/mpi.h"
#include <unistd.h>
#include <stdint.h>
#include <stdlib.h>
#include <unordered_map>


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

// This method determines the color of a ray going through the scene by tracing it through the scene and hitting objects.
// It has been modified to use CUDA as described below.
// Matching the C++ code would recurse enough into color() calls that
// it was blowing up the stack, so we have to turn this into a
// limited-depth loop instead.  Later code in the book limits to a max
// depth of 50, so we adapt this a few chapters early on the GPU.
__device__ vec3 color(const ray& r, hitable **world, hiprandState *local_rand_state) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0,1.0,1.0);
    for(int i = 0; i < 50; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            if(rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return vec3(0.0,0.0,0.0);
            }
        }
        else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f*(unit_direction.y() + 1.0f);
            vec3 c = (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return vec3(0.0,0.0,0.0); // exceeded recursion
}

__global__ void rand_init(hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state, int init_x, int init_y) {
    int i_local = threadIdx.x + blockIdx.x * blockDim.x;
    int j_local = threadIdx.y + blockIdx.y * blockDim.y;
    int i = i_local + init_x;
    int j = j_local + init_y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    int pixel_index_local = j_local*max_x + i_local;
    // Original: Each thread gets same seed, a different sequence number, no offset
    // hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
    // BUGFIX, see Issue#2: Each thread gets different seed, same sequence for
    // performance improvement of about 2x!
    hiprand_init(1984+pixel_index, 0, 0, &rand_state[pixel_index_local]);
}

__global__ void render(vec3 *frame_buffer, int max_x, int max_y, int ns, camera **cam, hitable **world, hiprandState *rand_state, int init_x, int init_y) {
    int i_local = threadIdx.x + blockIdx.x * blockDim.x + init_x;
    int j_local = threadIdx.y + blockIdx.y * blockDim.y + init_y;
    int i = i_local + init_x;
    int j = j_local + init_y;
    if((i >= max_x) || (j >= max_y)) return;
    // int pixel_index = j*max_x + i;
    int pixel_index_local = j_local*max_x + i_local;
    hiprandState local_rand_state = rand_state[pixel_index_local];
    vec3 col(0,0,0);
    for(int s=0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += color(r, world, &local_rand_state);
    }
    rand_state[pixel_index_local] = local_rand_state;
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    frame_buffer[pixel_index_local] = col;
}

__global__ void render_vec3_t(vec3_t *frame_buffer, int max_x, int max_y, int ns, camera **cam, hitable **world, hiprandState *rand_state, int init_x, int init_y) {
    int i_local = threadIdx.x + blockIdx.x * blockDim.x + init_x;
    int j_local = threadIdx.y + blockIdx.y * blockDim.y + init_y;
    int i = i_local + init_x;
    int j = j_local + init_y;
    if((i >= max_x) || (j >= max_y)) return;
    // int pixel_index = j*max_x + i;
    int pixel_index_local = j_local*max_x + i_local;
    hiprandState local_rand_state = rand_state[pixel_index_local];
    vec3_t col{0,0,0};
    for(int s=0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        auto col_addition = color(r, world, &local_rand_state);
        col.x += col_addition.x();
        col.y += col_addition.y();
        col.z += col_addition.z();
    }
    rand_state[pixel_index_local] = local_rand_state;
    col.x = sqrt(col.x / float(ns));
    col.y = sqrt(col.y / float(ns));
    col.z = sqrt(col.z / float(ns));
    frame_buffer[pixel_index_local] = col;
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(hitable **d_list, hitable **d_world, camera **d_camera, int nx, int ny, hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        d_list[0] = new sphere(vec3(0,-1000.0,-1), 1000,
                               new lambertian(vec3(0.5, 0.5, 0.5)));
        int i = 1;
        for(int a = -11; a < 11; a++) {
            for(int b = -11; b < 11; b++) {
                float choose_mat = RND;
                vec3 center(a+RND,0.2,b+RND);
                if(choose_mat < 0.8f) {
                    d_list[i++] = new sphere(center, 0.2,
                                             new lambertian(vec3(RND*RND, RND*RND, RND*RND)));
                }
                else if(choose_mat < 0.95f) {
                    d_list[i++] = new sphere(center, 0.2,
                                             new metal(vec3(0.5f*(1.0f+RND), 0.5f*(1.0f+RND), 0.5f*(1.0f+RND)), 0.5f*RND));
                }
                else {
                    d_list[i++] = new sphere(center, 0.2, new dielectric(1.5));
                }
            }
        }
        d_list[i++] = new sphere(vec3(0, 1,0),  1.0, new dielectric(1.5));
        d_list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
        d_list[i++] = new sphere(vec3(4, 1, 0),  1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));
        *rand_state = local_rand_state;
        *d_world  = new hitable_list(d_list, 22*22+1+3);

        vec3 lookfrom(13,2,3);
        vec3 lookat(0,0,0);
        float dist_to_focus = 10.0; (lookfrom-lookat).length();
        float aperture = 0.1;
        *d_camera   = new camera(lookfrom,
                                 lookat,
                                 vec3(0,1,0),
                                 30.0,
                                 float(nx)/float(ny),
                                 aperture,
                                 dist_to_focus);
    }
}

vec3 camera_move_vector() {
    return vec3(0,0,-0.1);
}

// Moves the camera's origin to create a new scene
__global__ void move_cam(camera **d_camera) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        (*d_camera)->origin += vec3(0,0,-0.1);
    }
}

__global__ void free_world(hitable **d_list, hitable **d_world, camera **d_camera) {
    for(int i=0; i < 22*22+1+3; i++) {
        delete ((sphere *)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete *d_world;
    delete *d_camera;
}

// Writes the image to a ppm file.
void write_frame_buffer(vec3 *frame_buffer, int nx, int ny, int max_x, int max_y) {
    FILE *f = fopen("output.ppm", "w");
    fprintf(f, "P3\n%d %d\n255\n", max_x, max_y);
    for(int j=0; j < max_y; j++) {
        for(int i=0; i < max_x; i++) {
            vec3 col = frame_buffer[j*max_x + i];
            int ir = int(255.99*col[0]);
            int ig = int(255.99*col[1]);
            int ib = int(255.99*col[2]);
            fprintf(f, "%d %d %d ", ir, ig, ib);
        }
        fprintf(f, "\n");
    }
    fclose(f);
}

// Writes the image to a ppm file.
void write_frame_buffer_arr(vec3 **frame_buffer_arr, int nx, int ny, int max_x, int max_y) {
    int size = sizeof frame_buffer_arr / sizeof frame_buffer_arr[0];
    int slice_height = max_y/size;
    FILE *f = fopen("output.ppm", "w");
    fprintf(f, "P3\n%d %d\n255\n", max_x, max_y);
    for(int j=0; j < max_y; j++) {
        for(int i=0; i < max_x; i++) {
            int slice = j/slice_height;
            vec3 col = frame_buffer_arr[slice][(j-(slice*slice_height))*max_x + i];
            int ir = int(255.99*col[0]);
            int ig = int(255.99*col[1]);
            int ib = int(255.99*col[2]);
            fprintf(f, "%d %d %d ", ir, ig, ib);
        }
        fprintf(f, "\n");
    }
    fclose(f);
}

// Renders a single image and writes it to a ppm file.
void test_render(int image_height, int image_width, int samples_per_pixel) {
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << image_width << "x" << image_height << " image with " << samples_per_pixel << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = image_width*image_height;
    size_t frame_buffer_size = num_pixels*sizeof(vec3);

    // allocate frame_buffer
    vec3 *frame_buffer;
    checkCudaErrors(hipMallocManaged((void **)&frame_buffer, frame_buffer_size));

    // allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));
    hiprandState *d_rand_state2;
    checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1*sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init<<<1,1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // make our world of hitables & the camera
    hitable **d_list;
    int num_hitables = 22*22+1+3;
    checkCudaErrors(hipMalloc((void **)&d_list, num_hitables*sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    create_world<<<1,1>>>(d_list, d_world, d_camera, image_width, image_height, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(image_width/tx+1,image_height/ty+1);
    dim3 threads(tx,ty);
    render_init<<<blocks, threads>>>(image_width, image_height, d_rand_state, 0, 0);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(frame_buffer, image_width, image_height, samples_per_pixel, d_camera, d_world, d_rand_state, 0, 0);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output frame_buffer as Image
    write_frame_buffer(frame_buffer, image_width, image_height, image_width, image_height);

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world,d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(frame_buffer));

    hipDeviceReset();
}

void benchmark_single(int image_height, int image_width, int samples_per_pixel, int num_frames_to_render) {
    int tx = 8;
    int ty = 8;

    std::cerr << "Benchmarking the rendering of " << image_width << "x" << image_height << " images with " << samples_per_pixel << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = image_width*image_height;
    size_t frame_buffer_size = num_pixels*sizeof(vec3);

    // allocate frame_buffer
    vec3 *frame_buffer;
    checkCudaErrors(hipMallocManaged((void **)&frame_buffer, frame_buffer_size));

    // allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));
    hiprandState *d_rand_state2;
    checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1*sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init<<<1,1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // make our world of hitables & the camera
    hitable **d_list;
    int num_hitables = 22*22+1+3;
    checkCudaErrors(hipMalloc((void **)&d_list, num_hitables*sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    create_world<<<1,1>>>(d_list, d_world, d_camera, image_width, image_height, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(image_width/tx+1,image_height/ty+1);
    dim3 threads(tx,ty);
    render_init<<<blocks, threads>>>(image_width, image_height, d_rand_state, 0, 0);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    for (int i = 0; i < num_frames_to_render; i++) {
        // Render the current frame and make sure it worked.
        render<<<blocks, threads>>>(frame_buffer, image_width, image_height, samples_per_pixel, d_camera, d_world, d_rand_state, 0, 0);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());
        // Move the camera to create the next frame.
        move_cam<<<blocks, threads>>>(d_camera);
    }
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds to generate " << num_frames_to_render << " frames.\n";
    std::cerr << "Average FPS: " << (double)num_frames_to_render / timer_seconds << "\n";

    // Output frame_buffer as Image
    write_frame_buffer(frame_buffer, image_width, image_height, image_width, image_height);

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world,d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(frame_buffer));

    hipDeviceReset();
}

void benchmark_tiled(int image_height, int image_width, int samples_per_pixel, int num_frames_to_render) {
    // int tx = 8;
    // int ty = 8;

    // std::cerr << "Benchmarking the rendering of " << image_width << "x" << image_height << " images with " << samples_per_pixel << " samples per pixel ";
    // std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    // ncclComm_t comms[4];

    // //managing 4 devices
    // int nDev = 4;
    // int devs[4] = { 0, 1, 2, 3 };

    // //allocating and initializing device buffers
    // hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*nDev);

    // int num_pixels = image_width*image_height;
    // size_t frame_buffer_size = num_pixels*sizeof(vec3);

    // int image_width_dev = image_width;
    // int image_height_dev = image_height / nDev;
    // if (image_width_dev * image_height_dev * nDev != image_height * image_width) {
    //     std::cerr << "Not an even split." << std::endl;
    //     exit(1);
    // }
    // int num_pixels_dev = num_pixels / nDev;
    // size_t frame_buffer_size_dev = frame_buffer_size / nDev;

    // vec3** frame_buffer = (vec3**)malloc(nDev * sizeof(vec3*));
    // hiprandState** d_rand_state = (hiprandState**)malloc(nDev * sizeof(hiprandState*));
    // hiprandState** d_rand_state2 = (hiprandState**)malloc(nDev * sizeof(hiprandState*));
    // hitable ***d_list = (hitable***)malloc(nDev * sizeof(hitable**));
    // int num_hitables = 22*22+1+3;
    // hitable ***d_world = (hitable***)malloc(nDev * sizeof(hitable**));
    // camera ***d_camera = (camera***)malloc(nDev * sizeof(camera**));

    // for (int i = 0; i < nDev; ++i) {
    //     checkCudaErrors(hipSetDevice(i));

    //     // allocate frame_buffer
    //     checkCudaErrors(hipMallocManaged((void **)&frame_buffer[i], frame_buffer_size_dev));

    //     // allocate random state
    //     checkCudaErrors(hipMalloc((void **)&d_rand_state[i], num_pixels_dev*sizeof(hiprandState)));
    //     checkCudaErrors(hipMalloc((void **)&d_rand_state2[i], 1*sizeof(hiprandState)));

    //     // we need that 2nd random state to be initialized for the world creation
    //     rand_init<<<1,1>>>(d_rand_state2[i]);
    //     checkCudaErrors(hipGetLastError());
    //     checkCudaErrors(hipDeviceSynchronize());

    //     // make our world of hitables & the camera
    //     checkCudaErrors(hipMalloc((void **)&d_list[i], num_hitables*sizeof(hitable *)));
    //     checkCudaErrors(hipMalloc((void **)&d_world[i], sizeof(hitable *)));
    //     checkCudaErrors(hipMalloc((void **)&d_camera[i], sizeof(camera *)));
    //     create_world<<<1,1>>>(d_list[i], d_world[i], d_camera[i], image_width_dev, image_height_dev, d_rand_state2[i]);
    //     checkCudaErrors(hipGetLastError());
    //     checkCudaErrors(hipDeviceSynchronize());

    //     checkCudaErrors(hipStreamCreate(s+i));
    // }

    // //initializing NCCL
    // NCCLCHECK(ncclCommInitAll(comms, nDev, devs));

    // clock_t start, stop;
    // start = clock();

    // dim3 blocks(image_width_dev/tx+1,image_height_dev/ty+1);
    // dim3 threads(tx,ty);
    // for (int i = 0; i < nDev; ++i) {
    //     checkCudaErrors(hipSetDevice(i));
    //     // Render our buffer
    //     render_init<<<blocks, threads>>>(image_width_dev, image_height_dev, d_rand_state[i], 0, i*image_height_dev);
    // }

    // //synchronizing on CUDA streams to wait for completion of NCCL operation
    // for (int i = 0; i < nDev; ++i) {
    //     checkCudaErrors(hipSetDevice(i));
    //     checkCudaErrors(hipGetLastError());
    //     checkCudaErrors(hipStreamSynchronize(s[i]));
    // }

    // for (int f = 0; f < num_frames_to_render; f++) {
    //     for (int i = 0; i < nDev; ++i) {
    //         checkCudaErrors(hipSetDevice(i));
    //         // Render the current frame and make sure it worked.
    //         render<<<blocks, threads>>>(frame_buffer[i], image_width_dev, image_height_dev, samples_per_pixel, d_camera[i], d_world[i], d_rand_state[i], 0, i*image_height_dev);
    //     }

    //     //synchronizing on CUDA streams to wait for completion of NCCL operation
    //     for (int i = 0; i < nDev; ++i) {
    //         checkCudaErrors(hipSetDevice(i));
    //         checkCudaErrors(hipGetLastError());
    //         checkCudaErrors(hipStreamSynchronize(s[i]));
    //     }

    //     for (int i = 0; i < nDev; ++i) {
    //         checkCudaErrors(hipSetDevice(i));
    //         // Move the camera to create the next frame.
    //         move_cam<<<blocks, threads>>>(d_camera[i]);
    //     }
    // }

    // stop = clock();
    // double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    // std::cerr << "took " << timer_seconds << " seconds to generate " << num_frames_to_render << " frames.\n";
    // std::cerr << "Average FPS: " << (double)num_frames_to_render / timer_seconds << "\n";

    // // Output frame_buffer as Image
    // write_frame_buffer_arr(frame_buffer, image_width, image_height, image_width, image_height);

    // //free device buffers
    // for (int i = 0; i < nDev; ++i) {
    //     checkCudaErrors(hipSetDevice(i));
    //     // clean up
    //     checkCudaErrors(hipDeviceSynchronize());
    //     free_world<<<1,1>>>(d_list[i],d_world[i],d_camera[i]);
    //     checkCudaErrors(hipGetLastError());
    //     checkCudaErrors(hipFree(d_camera[i]));
    //     checkCudaErrors(hipFree(d_world[i]));
    //     checkCudaErrors(hipFree(d_list[i]));
    //     checkCudaErrors(hipFree(d_rand_state[i]));
    //     checkCudaErrors(hipFree(d_rand_state2[i]));
    //     checkCudaErrors(hipFree(frame_buffer[i]));
    // }

    // //finalizing NCCL
    // for(int i = 0; i < nDev; ++i)
    //     ncclCommDestroy(comms[i]);

    // printf("Success \n");

    // hipDeviceReset();
    (void*)0;
}

MPI_Datatype VEC3;

// Initializes the camera origins data structure.
__global__ void init_origins_for_frames(vec3_t* camera_origins_for_frames, int num_frames_to_render, camera** d_camera) {
    auto current_origin = (*d_camera)->origin;
    auto move_vector = vec3(0,0,-0.1);
    camera_origins_for_frames[0] = {current_origin.x(), current_origin.y(), current_origin.z()};
    for (int i = 1; i < num_frames_to_render; ++i) {
        // Each subsequent frame is offset by the move vector.
        camera_origins_for_frames[i].x = camera_origins_for_frames[i-1].x + move_vector.x();
        camera_origins_for_frames[i].y = camera_origins_for_frames[i-1].y + move_vector.y();
        camera_origins_for_frames[i].z = camera_origins_for_frames[i-1].z + move_vector.z();
    }
}

// Moves camera origin based on input vector.
__global__ void update_camera_origin(vec3_t* new_origin, camera** d_camera) {
    (*d_camera)->origin.e[0] = new_origin->x;
    (*d_camera)->origin.e[1] = new_origin->y;
    (*d_camera)->origin.e[2] = new_origin->z;
}


void benchmark_frame(int argc, char **argv, int image_height, int image_width, int samples_per_pixel, int num_frames_to_render, int network_latency_in_us, bool has_stragglers) {
    std::cerr << "Benchmarking the rendering of " << num_frames_to_render << " " << image_width << "x" << image_height << " images with " << samples_per_pixel << " samples per pixel " << std::endl;

    int num_pixels = image_width*image_height;
    size_t frame_buffer_size = num_pixels*sizeof(vec3_t)*num_frames_to_render;

    // Initialize MPI.
    int num_procs, rank = 0;
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &num_procs);

    std::cout << "Running on " << num_procs << " processes" << std::endl;

    // Create MPI Vec3 Type.
    const int nitems = 3;
    int blocklengths[3] = {1, 1, 1};
    MPI_Datatype types[7] = {MPI_FLOAT, MPI_FLOAT, MPI_FLOAT};
    MPI_Aint offsets[3];
    offsets[0] = offsetof(vec3_t, x);
    offsets[1] = offsetof(vec3_t, y);
    offsets[2] = offsetof(vec3_t, z);
    MPI_Type_create_struct(nitems, blocklengths, offsets, types, &VEC3);
    MPI_Type_commit(&VEC3);

    // Allocate random state.
    hiprandState *d_rand_state;
    hiprandState *d_rand_state2;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));
    checkCudaErrors(hipMalloc((void **)&d_rand_state2, sizeof(hiprandState)));

    // Allocate world state.
    int num_hitables = 22*22+1+3;
    hitable **d_list;
    hitable **d_world;
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_list, num_hitables*sizeof(hitable *)));
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));

    // Generate and broadcast random state from rank 0 to all ranks.
    if (rank == 0) {
        // Generate.
        rand_init<<<1,1>>>(d_rand_state2);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());
    }

    // Broadcast.
    MPI_Bcast((void *)d_rand_state2, sizeof(hiprandState), MPI_BYTE, 0, MPI_COMM_WORLD);

    // All ranks create world.
    create_world<<<1,1>>>(d_list, d_world, d_camera, image_width, image_height, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // All ranks initialize rendering.
    int tx = 8;
    int ty = 8;
    dim3 blocks(image_width/tx+1,image_height/ty+1);
    dim3 threads(tx,ty);
    render_init<<<blocks, threads>>>(image_width, image_height, d_rand_state, 0, 0);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();

    // Now we begin rendering pipeline.
    // Goal is to mimic camera moving through the scene at a defined rate.
    if (rank == 0) {
        // Allocate frame buffer.
        vec3_t *frame_buffer;
        checkCudaErrors(hipMallocManaged((void **)&frame_buffer, frame_buffer_size));

        // Initialize camera origins for each frame.
        vec3_t* camera_origins_for_frames;
        checkCudaErrors(hipMalloc((void **)&camera_origins_for_frames, num_frames_to_render * sizeof(vec3_t)));
        init_origins_for_frames<<<1,1>>>(camera_origins_for_frames, num_frames_to_render, d_camera);

        // Now we begin our work assignment.
        std::vector<int> free_gpus;
        for (int i = 0; i < num_procs; i++) {
            free_gpus.push_back(i);
        }
        std::vector<int> remaining_frames;
        for (int i = 0; i < num_frames_to_render; i++) {
            remaining_frames.push_back(i);
        }
        std::unordered_map<int, int> work_assignment;
        std::unordered_map<int, MPI_Request*> work_requests;
        while (remaining_frames.size() > 0) {
            // Assignment loop.
            for (auto frame: remaining_frames) {
                if (work_assignment.find(frame) == work_assignment.end()) {
                    // We have not assigned this frame yet.
                    // Pick a gpu and assign it to this frame.
                    if (free_gpus.size() == 0) {
                        // No more free gpus.
                        break;
                    }
                    // Grab the first free gpu and remove it.
                    int gpu = free_gpus.back();
                    work_assignment[frame] = gpu;
                    free_gpus.pop_back();
                    if (gpu != rank) {
                        // This is a remote GPU, send to be completed via MPI.
                        work_requests[frame] = new MPI_Request();
                        MPI_Send(&camera_origins_for_frames[frame], 1, VEC3, gpu, 0, MPI_COMM_WORLD);
                        MPI_Irecv(frame_buffer + frame * num_pixels, num_pixels, VEC3, gpu, 0, MPI_COMM_WORLD, work_requests[frame]);
                    }
                }
            }

            // Check for assignment completions.
            for (auto frame_gpu: work_assignment) {
                auto frame_id = frame_gpu.first;
                auto gpu = frame_gpu.second;
                if (frame_gpu.second == rank) {
                    // This is our work. We can now render it.

                    // Render the current frame and make sure it worked.
                    render_vec3_t<<<blocks, threads>>>(frame_buffer + frame_id * num_pixels, image_width, image_height, samples_per_pixel, d_camera, d_world, d_rand_state, 0, 0);
                    checkCudaErrors(hipGetLastError());
                    checkCudaErrors(hipDeviceSynchronize());

                    // Put GPU back in free list.
                    free_gpus.push_back(gpu);
                    // Remove frame from remaining frames.
                    remaining_frames.erase(std::remove(remaining_frames.begin(), remaining_frames.end(), frame_id), remaining_frames.end());
                    // Remove frame from work assignment.
                    work_assignment.erase(frame_id);
                } else {
                    // This is remote work, poll for completion.
                    int completed;
                    MPI_Test(work_requests[frame_id], &completed, MPI_STATUS_IGNORE);
                    if (completed) {
                        // This is done.
                        // Remove frame from remaining frames.
                        remaining_frames.erase(std::remove(remaining_frames.begin(), remaining_frames.end(), frame_id), remaining_frames.end());
                        // Put GPU back in free list.
                        free_gpus.push_back(gpu);
                        // Remove frame from work assignment.
                        work_assignment.erase(frame_id);
                    }
                }
            }

        }

    } else {
        // Allocate frame buffer.
        vec3_t *frame_buffer;
        checkCudaErrors(hipMallocManaged((void **)&frame_buffer, frame_buffer_size / num_frames_to_render));

        // Stores the camera origins from the root.
        vec3_t camera_origin = {0,0,0};
        while(true) {
            MPI_Status status;
            // Attempt to receive the camera origin for the frame.
            MPI_Recv(&camera_origin, 1, VEC3, 0, 0, MPI_COMM_WORLD, &status);
            // Update camera origin for the frame.
            update_camera_origin<<<1,1>>>(&camera_origin, d_camera);
            // Render the frame.
            render_vec3_t<<<blocks, threads>>>(frame_buffer, image_width, image_height, samples_per_pixel, d_camera, d_world, d_rand_state, 0, 0);
            checkCudaErrors(hipGetLastError());
            checkCudaErrors(hipDeviceSynchronize());

            // Send the frame back to the root.
            MPI_Send(frame_buffer, num_pixels, VEC3, 0, 0, MPI_COMM_WORLD);
        }
    }

    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds to generate " << num_frames_to_render << " frames.\n";
    std::cerr << "Average FPS: " << (double)num_frames_to_render / timer_seconds << "\n";

    MPI_Abort(MPI_COMM_WORLD, 0);
    MPI_Finalize();

    printf("Success \n");

    hipDeviceReset();
}

// Benchmarks the throughput of a rendering type.
void benchmark_rendering(std::string rendering_strategy, int image_height, int image_width, int samples_per_pixel, int num_frames_to_render, int network_latency_in_us, bool has_stragglers, int argc, char **argv) {
    if (strcmp(rendering_strategy.c_str(), "singlenode") == 0) {
        benchmark_single(image_height, image_width, samples_per_pixel, num_frames_to_render);
    } else if (strcmp(rendering_strategy.c_str(), "tiled") == 0) {
        benchmark_tiled(image_height, image_width, samples_per_pixel, num_frames_to_render);
    } else if (strcmp(rendering_strategy.c_str(), "frame") == 0) {
        benchmark_frame(argc, argv, image_height, image_width, samples_per_pixel, num_frames_to_render, network_latency_in_us, has_stragglers);
    }
}

int main(int argc, char **argv) {
    // Parse Args
    if (find_arg_idx(argc, argv, "-h") >= 0) {
        //std::cout << "Options:" << std::endl;
        //std::cout << "-h: see this help" << std::endl;
        //std::cout << "-t <int>: type, 0 = test, 1 = benchmark" << std::endl;
        //std::cout << "-r <rendering strategy>: singlenode/tiled/frame" << std::endl;
        //std::cout << "-v <int>: vertical height of image in pixels" << std::endl;
        //std::cout << "-w <int>: width of image in pixels" << std::endl;
        //std::cout << "-s <int>: number of samples per pixel" << std::endl;
        //std::cout << "-f <int>: number of frames to render" << std::endl;
        //std::cout << "-l <int>: emulated network latency" << std::endl;
        //std::cout << "-a <int>: stragglers, 0 = no, 1 = yes" << std::endl;
        return 0;
    }

    int image_height = find_int_arg(argc, argv, "-v", 800);
    int image_width = find_int_arg(argc, argv, "-w", 1200);
    int samples_per_pixel = find_int_arg(argc, argv, "-s", 10);

    int type = find_int_arg(argc, argv, "-t", 0);
    if (type == 0) {
        test_render(image_height, image_width, samples_per_pixel);
        return 0;
    }

    int num_frames_to_render = find_int_arg(argc, argv, "-f", 30);

    std::string rendering_strategy = find_string_option(argc, argv, "-r", std::string("singlenode"));
    int network_latency_in_us = find_int_arg(argc, argv, "0", 0);
    bool has_stragglers = find_int_arg(argc, argv, "-a", 0) == 1; 
    if (strcmp(rendering_strategy.c_str(), "singlenode") != 0  && strcmp(rendering_strategy.c_str(), "tiled") != 0 && strcmp(rendering_strategy.c_str(), "frame") != 0) {
        std::cerr << "Unknown rendering strategy: " << rendering_strategy << std::endl;
        return 1;
    }

    benchmark_rendering(rendering_strategy, image_height, image_width, samples_per_pixel, num_frames_to_render, network_latency_in_us, has_stragglers, argc, argv);
}