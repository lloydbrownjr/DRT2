#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <time.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>
#include "../common/vec3.h"
#include "../common/ray.h"
#include "sphere.h"
#include "hitable_list.h"
#include "camera.h"
#include "material.h"
#include "cuda_errors.h"
#include "../common/options.h"
#include <vector>

#define num_hitables (22*22 + 1 + 3)

// This method determines the color of a ray going through the scene by tracing it through the scene and hitting objects.
// It has been modified to use CUDA as described below.
// Matching the C++ code would recurse enough into color() calls that
// it was blowing up the stack, so we have to turn this into a
// limited-depth loop instead.  Later code in the book limits to a max
// depth of 50, so we adapt this a few chapters early on the GPU.
__device__ vec3 color(const ray& r, hitable **world, hiprandState *local_rand_state) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0,1.0,1.0);
    for(int i = 0; i < 50; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            if(rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return vec3(0.0, 0.0, 0.0);
            }
        }
        else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f * (unit_direction.y() + 1.0f);
            vec3 c = (1.0f-t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return vec3(0.0, 0.0, 0.0); // exceeded recursion
}

__global__ void rand_init(hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void render_init_tiled(int x_range, int y_range, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i >= x_range || j >= y_range) {
        return;
    }

    int pixel_index = j * x_range + i;
    // Original: Each thread gets same seed, a different sequence number, no offset
    // hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
    // BUGFIX, see Issue#2: Each thread gets different seed, same sequence for
    // performance improvement of about 2x!
    hiprand_init(1984+pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render_tiled(vec3 *frame_buffer, int image_width, int image_height, int x_start, int y_start, int x_range, int y_range,
        int number_samples, camera **cam, hitable **world, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= x_range || j >= y_range) {
        return;
    }
    int local_index = j * x_range + i;
    hiprandState local_rand_state = rand_state[local_index];
    int pixel_x = x_start + i;
    int pixel_y = y_start + j;
    int pixel_index = pixel_y * image_width + pixel_x;
    vec3 col(0, 0, 0);
    for(int sample = 0; sample < number_samples; sample++) {
        float u = float(pixel_x + hiprand_uniform(&local_rand_state)) / float(image_width);
        float v = float(pixel_y + hiprand_uniform(&local_rand_state)) / float(image_height);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += color(r, world, &local_rand_state);
    }
    rand_state[local_index] = local_rand_state;
    col /= float(number_samples);
    frame_buffer[pixel_index] = col.getsqrt();
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(hitable **d_list, hitable **d_world, camera **d_camera, int nx, int ny, hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        d_list[0] = new sphere(vec3(0, -1000, -1), 1000,
                               new lambertian(vec3(0.5, 0.5, 0.5)));
        int i = 1;
        for(int a = -11; a < 11; a++) {
            for(int b = -11; b < 11; b++) {
                float choose_mat = RND;
                vec3 center(a+RND,0.2,b+RND);
                if (choose_mat < 0.8f) {
                    d_list[i++] = new sphere(center, 0.2,
                                             new lambertian(vec3(RND*RND, RND*RND, RND*RND)));
                }
                else if (choose_mat < 0.95f) {
                    d_list[i++] = new sphere(center, 0.2,
                                             new metal(vec3(0.5f*(1.0f+RND), 0.5f*(1.0f+RND), 0.5f*(1.0f+RND)), 0.5f*RND));
                }
                else {
                    d_list[i++] = new sphere(center, 0.2, new dielectric(1.5));
                }
            }
        }
        d_list[i++] = new sphere(vec3(0, 1,0),  1.0, new dielectric(1.5));
        d_list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
        d_list[i++] = new sphere(vec3(4, 1, 0),  1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));
        *rand_state = local_rand_state;
        *d_world  = new hitable_list(d_list, num_hitables);

        vec3 lookfrom(13, 2, 3);
        vec3 lookat(0,0,0);
        float dist_to_focus = 10.0; (lookfrom-lookat).length();
        float aperture = 0.1;
        *d_camera   = new camera(lookfrom,
                                 lookat,
                                 vec3(0,1,0),
                                 30.0,
                                 float(nx)/float(ny),
                                 aperture,
                                 dist_to_focus);
    }
}

// Moves the camera's origin to create a new scene
__global__ void move_cam(camera **d_camera, int steps = 1) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        (*d_camera)->origin += steps * vec3(0, 0, -0.1);
    }
}

__global__ void free_world(hitable **d_list, hitable **d_world, camera **d_camera) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        for (int i = 0; i < num_hitables; i++) {
            delete ((sphere *)d_list[i])->mat_ptr;
            delete d_list[i];
        }
        delete *d_world;
        delete *d_camera;
    }
}

// Writes the image to a ppm file.
void write_frame_buffer(vec3 *frame_buffer, int nx, int ny, int max_x, int max_y) {
    FILE *f = fopen("output.ppm", "w");
    fprintf(f, "P3\n%d %d\n255\n", max_x, max_y);
    for(int j = 0; j < max_y; j++) {
        for(int i = 0; i < max_x; i++) {
            vec3 col = frame_buffer[j*max_x + i];
            int ir = int(255.99 * col[0]);
            int ig = int(255.99 * col[1]);
            int ib = int(255.99 * col[2]);
            fprintf(f, "%d %d %d ", ir, ig, ib);
        }
        fprintf(f, "\n");
    }
    fclose(f);
}

// Renders a single image and writes it to a ppm file.
void test_render(int image_height, int image_width, int samples_per_pixel) {
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << image_width << "x" << image_height << " image with " << samples_per_pixel << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = image_width*image_height;
    size_t frame_buffer_size = num_pixels*sizeof(vec3);

    // allocate frame_buffer
    vec3 *frame_buffer;
    checkCudaErrors(hipMallocManaged((void **)&frame_buffer, frame_buffer_size));

    // allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));
    hiprandState *d_rand_state2;
    checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1*sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init<<<1, 1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // make our world of hitables & the camera
    hitable **d_list;
    checkCudaErrors(hipMalloc((void **)&d_list, num_hitables*sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    create_world<<<1, 1>>>(d_list, d_world, d_camera, image_width, image_height, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(image_width/tx+1,image_height/ty+1);
    dim3 threads(tx,ty);
    // render_init<<<blocks, threads>>>(image_width, image_height, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    // render<<<blocks, threads>>>(frame_buffer, image_width, image_height, samples_per_pixel, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output frame_buffer as Image
    write_frame_buffer(frame_buffer, image_width, image_height, image_width, image_height);

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world,d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(frame_buffer));

    hipDeviceReset();
}

void benchmark_single(int image_height, int image_width, int samples_per_pixel, int num_frames_to_render) {
    int tx = 8;
    int ty = 8;

    std::cerr << "Benchmarking the rendering of " << image_width << "x" << image_height << " images with " << samples_per_pixel << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = image_width*image_height;
    size_t frame_buffer_size = num_pixels*sizeof(vec3);

    // allocate frame_buffer
    vec3 *frame_buffer;
    checkCudaErrors(hipMallocManaged((void **)&frame_buffer, frame_buffer_size));

    // allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));
    hiprandState *d_rand_state2;
    checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1*sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init<<<1,1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // make our world of hitables & the camera
    hitable **d_list;
    checkCudaErrors(hipMalloc((void **)&d_list, num_hitables * sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    create_world<<<1,1>>>(d_list, d_world, d_camera, image_width, image_height, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(image_width/tx+1,image_height/ty+1);
    dim3 threads(tx,ty);
    render_init_tiled<<<blocks, threads>>>(image_width, image_height, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    for (int i = 0; i < num_frames_to_render; i++) {
        // Render the current frame and make sure it worked.
        render_tiled<<<blocks, threads>>>(frame_buffer, image_width, image_height, 0, 0, image_width, image_height, samples_per_pixel, d_camera, d_world, d_rand_state);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());
        // Move the camera to create the next frame.
        move_cam<<<blocks, threads>>>(d_camera);
    }
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds to generate " << num_frames_to_render << " frames.\n";
    std::cerr << "Average FPS: " << (double)num_frames_to_render / timer_seconds << "\n";

    // Output frame_buffer as Image
    write_frame_buffer(frame_buffer, image_width, image_height, image_width, image_height);

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world,d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(frame_buffer));

    hipDeviceReset();
}

void benchmark_tiled(int image_height, int image_width, int samples_per_pixel, int num_frames_to_render, int num_gpus = -1) {
    int tx = 8;
    int ty = 8;

    std::cerr << "Benchmarking the rendering of " << image_width << "x" << image_height << " images with " << samples_per_pixel << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = image_width * image_height;

    // allocate frame_buffer
    vec3 *frame_buffer;
    checkCudaErrors(hipMallocManaged((void **)&frame_buffer, num_pixels * sizeof(vec3)));

    int available_gpus;
    checkCudaErrors(hipGetDeviceCount(&available_gpus));
    if (num_gpus > available_gpus) {
        std::cerr << "requeted more than available GPUs, capping." << std::endl;
        num_gpus = available_gpus;
    } else if (num_gpus == -1) {
        num_gpus = available_gpus;
    }

    int num_streams = num_gpus;
    
    int per_gpu_width = image_width;
    int per_gpu_height = image_height / num_gpus;
    int num_pixels_per_gpu = per_gpu_width * per_gpu_height;

    dim3 blocks(per_gpu_width / tx + 1, per_gpu_height / ty + 1);
    dim3 threads(tx, ty);

    // allocate random state
    using std::vector;
    vector<hiprandState *> d_rand_state(num_gpus);
    vector<hiprandState *> d_rand_state2(num_gpus);
    vector<hipStream_t> streams(num_streams);
    vector<hitable **> d_list(num_gpus);
    vector<hitable **> d_world(num_gpus);
    vector<camera **> d_camera(num_gpus);
    for (int gpu_id = 0; gpu_id < num_gpus; gpu_id++) {
        checkCudaErrors(hipSetDevice(gpu_id));
        checkCudaErrors(hipMalloc((void **) &d_rand_state[gpu_id], num_pixels_per_gpu * sizeof(hiprandState)));
        checkCudaErrors(hipMalloc((void **) &d_rand_state2[gpu_id], 1 * sizeof(hiprandState)));
        checkCudaErrors(hipStreamCreate(&streams[gpu_id]));
        rand_init<<<1, 1>>>(d_rand_state2[gpu_id]);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipMalloc((void **) &d_list[gpu_id], num_hitables * sizeof(hitable *)));
        checkCudaErrors(hipMalloc((void **) &d_world[gpu_id], 1 * sizeof(hitable *)));
        checkCudaErrors(hipMalloc((void **) &d_camera[gpu_id], 1 * sizeof(camera *)));
        hipDeviceSynchronize();
        create_world<<<1, 1>>>(d_list[gpu_id], d_world[gpu_id], d_camera[gpu_id], image_width, image_height, d_rand_state2[gpu_id]);
        checkCudaErrors(hipGetLastError());
    }

    clock_t start, stop;
    start = clock();
    // Render our buffer
    for (int gpu_id = 0; gpu_id < num_gpus; gpu_id++) {
        checkCudaErrors(hipSetDevice(gpu_id));
        render_init_tiled<<<blocks, threads, 0, streams[gpu_id]>>>(per_gpu_width, per_gpu_height, d_rand_state[gpu_id]);
        // render_init<<<blocks, threads, 0, streams[gpu_id]>>>(image_width, image_height, d_rand_state[gpu_id], gpu_id, num_gpus);
    }
    for (int stream_id = 0; stream_id < num_streams; stream_id++) {
        checkCudaErrors(hipStreamSynchronize(streams[stream_id]));
    }
    for (int i = 0; i < num_frames_to_render; i++) {
        for (int gpu_id = 0; gpu_id < num_gpus; gpu_id++) {
            checkCudaErrors(hipSetDevice(gpu_id));
            int x_start = 0, y_start = image_height * gpu_id / num_gpus;
            render_tiled<<<blocks, threads, 0, streams[gpu_id]>>>(frame_buffer, image_width, image_height, x_start, y_start, per_gpu_width, per_gpu_height,
                samples_per_pixel, d_camera[gpu_id], d_world[gpu_id], d_rand_state[gpu_id]);
            checkCudaErrors(hipGetLastError());
            move_cam<<<1, 1, 0, streams[gpu_id]>>>(d_camera[gpu_id]);
            checkCudaErrors(hipGetLastError());
        }
        for (int gpu_id = 0; gpu_id < num_gpus; gpu_id++) {
            checkCudaErrors(hipStreamSynchronize(streams[gpu_id]));
        }
    }

    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds to generate " << num_frames_to_render << " frames.\n";
    std::cerr << "Average FPS: " << (double)num_frames_to_render / timer_seconds << "\n";

    // Output frame_buffer as Image
    write_frame_buffer(frame_buffer, image_width, image_height, image_width, image_height);

    // clean up
    for (int gpu_id = 0; gpu_id < num_gpus; gpu_id++) {
        checkCudaErrors(hipSetDevice(gpu_id));
        free_world<<<1, 1>>>(d_list[gpu_id], d_world[gpu_id], d_camera[gpu_id]);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipFree(d_camera[gpu_id]));
        checkCudaErrors(hipFree(d_world[gpu_id]));
        checkCudaErrors(hipFree(d_list[gpu_id]));
        checkCudaErrors(hipStreamDestroy(streams[gpu_id]));
        checkCudaErrors(hipFree(d_rand_state2[gpu_id]));
        checkCudaErrors(hipFree(d_rand_state[gpu_id]));
    }
    checkCudaErrors(hipFree(frame_buffer));

    hipDeviceReset();
}

void benchmark_frame(int image_height, int image_width, int samples_per_pixel, int num_frames_to_render) {
    std::cerr << "Not implemented." << std::endl;
    exit(1);
}

// Benchmarks the throughput of a rendering type.
void benchmark_rendering(std::string rendering_strategy, int image_height, int image_width, int samples_per_pixel, int num_frames_to_render, int requested_gpus) {
    if (strcmp(rendering_strategy.c_str(), "singlenode") == 0) {
        benchmark_single(image_height, image_width, samples_per_pixel, num_frames_to_render);
    } else if (strcmp(rendering_strategy.c_str(), "tiled") == 0) {
        benchmark_tiled(image_height, image_width, samples_per_pixel, num_frames_to_render, requested_gpus);
    } else if (strcmp(rendering_strategy.c_str(), "frame") == 0) {
        benchmark_frame(image_height, image_width, samples_per_pixel, num_frames_to_render);
    }
}

int main(int argc, char **argv) {
    // Parse Args
    if (find_arg_idx(argc, argv, "-h") >= 0) {
        std::cout << "Options:" << std::endl;
        std::cout << "-h: see this help" << std::endl;
        std::cout << "-t <int>: type, 0 = test, 1 = benchmark" << std::endl;
        std::cout << "-r <rendering strategy>: singlenode/tiled/frame" << std::endl;
        std::cout << "-v <int>: vertical height of image in pixels" << std::endl;
        std::cout << "-w <int>: width of image in pixels" << std::endl;
        std::cout << "-s <int>: number of samples per pixel" << std::endl;
        std::cout << "-f <int>: number of frames to render" << std::endl;
        std::cout << "-g <int>: number of gpus to use" << std::endl;
        return 0;
    }

    int image_height = find_int_arg(argc, argv, "-v", 800);
    int image_width = find_int_arg(argc, argv, "-w", 1200);
    int samples_per_pixel = find_int_arg(argc, argv, "-s", 10);

    int type = find_int_arg(argc, argv, "-t", 0);
    if (type == 0) {
        test_render(image_height, image_width, samples_per_pixel);
        return 0;
    }

    int num_frames_to_render = find_int_arg(argc, argv, "-f", 30);
    int requested_gpus = find_int_arg(argc, argv, "-g", -1);

    std::string rendering_strategy = find_string_option(argc, argv, "-r", std::string("singlenode"));
    if (strcmp(rendering_strategy.c_str(), "singlenode") != 0  && strcmp(rendering_strategy.c_str(), "tiled") != 0 && strcmp(rendering_strategy.c_str(), "frame") != 0) {
        std::cerr << "Unknown rendering strategy: " << rendering_strategy << std::endl;
        return 1;
    }

    benchmark_rendering(rendering_strategy, image_height, image_width, samples_per_pixel, num_frames_to_render, requested_gpus);
}