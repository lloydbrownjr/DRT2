#include "hip/hip_runtime.h"
#include <float.h>
#include <iostream>
#include <map>
#include <string>
#include <time.h>
#include <unordered_map>
#include <unordered_set>
#include <vector>
#include <unistd.h>

#include <hiprand/hiprand_kernel.h>
#include <mpi.h>

#include "../common/options.h"
#include "../common/ray.h"
#include "../common/vec3.h"
#include "camera.h"
#include "cuda_errors.h"
#include "hitable_list.h"
#include "material.h"
#include "sphere.h"

static int num_hitables = (22*22 + 1 + 3);

#define NORMAL 0
#define UNBALANCED 1
#define BALANCED 2

// This method determines the color of a ray going through the scene by tracing it through the scene and hitting objects.
// It has been modified to use CUDA as described below.
// Matching the C++ code would recurse enough into color() calls that
// it was blowing up the stack, so we have to turn this into a
// limited-depth loop instead.  Later code in the book limits to a max
// depth of 50, so we adapt this a few chapters early on the GPU.
__device__ vec3 color(const ray& r, hitable **world, hiprandState *local_rand_state) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0,1.0,1.0);
    for(int i = 0; i < 50; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            if(rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return vec3(0.0, 0.0, 0.0);
            }
        }
        else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f * (unit_direction.y() + 1.0f);
            vec3 c = (1.0f-t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return vec3(0.0, 0.0, 0.0); // exceeded recursion
}

__global__ void rand_init(hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void render_init_tiled(int x_range, int y_range, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i >= x_range || j >= y_range) {
        return;
    }

    int pixel_index = j * x_range + i;
    // Original: Each thread gets same seed, a different sequence number, no offset
    // hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
    // BUGFIX, see Issue#2: Each thread gets different seed, same sequence for
    // performance improvement of about 2x!
    hiprand_init(1984+pixel_index, 0, 0, &rand_state[pixel_index]);
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void render_tiled(vec3 *frame_buffer, int image_width, int image_height, int x_start, int y_start, int x_range, int y_range,
        int number_samples, camera **cam, hitable **world, hiprandState *rand_state, int straggler = 0) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= x_range || j >= y_range) {
        return;
    }
    int local_index = j * x_range + i;
    hiprandState local_rand_state = rand_state[local_index];
    int pixel_x = x_start + i;
    int pixel_y = y_start + j;
    int pixel_index = pixel_y * image_width + pixel_x;
    vec3 col(0, 0, 0);
    for(int sample = 0; sample < number_samples; sample++) {
        float u = float(pixel_x + hiprand_uniform(&local_rand_state)) / float(image_width);
        float v = float(pixel_y + hiprand_uniform(&local_rand_state)) / float(image_height);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += color(r, world, &local_rand_state);
    }
    rand_state[local_index] = local_rand_state;
    col /= float(number_samples);
    frame_buffer[pixel_index] = col.getsqrt();
    if (straggler == 1) {
        if (RND > 0.5 == 0) {
            clock_t start_clock = clock();
            clock_t clock_offset = 0;
            clock_t clock_count = 2.19 * pow(10, 9) * 2; // in clock cycles with 2.19 * 10^9 Hz, 2 sec delay
            while (clock_offset < clock_count)
            {
                clock_offset = clock() - start_clock;
            }
        }
    }
}

__global__ void create_world(hitable **d_list, hitable **d_world, camera **d_camera, int nx, int ny, hiprandState *rand_state, int num_objs) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        d_list[0] = new sphere(vec3(0, -1000, -1), 1000,
                               new lambertian(vec3(0.5, 0.5, 0.5)));
        int i = 1;
        for(int a = -11; a < 11; a++) {
            for(int b = -11; b < 11; b++) {
                float choose_mat = RND;
                vec3 center(a+RND,0.2,b+RND);
                if (choose_mat < 0.8f) {
                    d_list[i++] = new sphere(center, 0.2,
                                             new lambertian(vec3(RND*RND, RND*RND, RND*RND)));
                }
                else if (choose_mat < 0.95f) {
                    d_list[i++] = new sphere(center, 0.2,
                                             new metal(vec3(0.5f*(1.0f+RND), 0.5f*(1.0f+RND), 0.5f*(1.0f+RND)), 0.5f*RND));
                }
                else {
                    d_list[i++] = new sphere(center, 0.2, new dielectric(1.5));
                }
            }
        }
        d_list[i++] = new sphere(vec3(0, 1,0),  1.0, new dielectric(1.5));
        d_list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
        d_list[i++] = new sphere(vec3(4, 1, 0),  1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));
        *rand_state = local_rand_state;
        *d_world  = new hitable_list(d_list, num_objs);

        vec3 lookfrom(13, 2, 3);
        vec3 lookat(0,0,0);
        float dist_to_focus = 10.0; (lookfrom-lookat).length();
        float aperture = 0.1;
        *d_camera   = new camera(lookfrom,
                                 lookat,
                                 vec3(0,1,0),
                                 30.0,
                                 float(nx)/float(ny),
                                 aperture,
                                 dist_to_focus);
    }
}

// The goal of this world creation is to make a world that is completely load balanced.
__global__ void create_world_balanced(hitable **d_list, hitable **d_world, camera **d_camera, int nx, int ny, hiprandState *rand_state, int num_objs) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        int i = 0;
        int xtrans = 35;
        int width = (int)sqrt((float)num_objs);
        int xspace = 2;
        for(int a = -width*xspace + xtrans; a < width*xspace + xtrans; a+=xspace) {
            for(int b = -width; b < width; b++) {
                float choose_mat = RND;
                vec3 center(a, b, 0.2);
                if (i % 2 == 0) {
                    d_list[i++] = new sphere(center, 1, new dielectric(1.5));
                } else {
                    d_list[i++] = new sphere(center, 1, new metal(vec3(0.7, 0.6, 0.5), 0.0));
                }
            }
        }
        *rand_state = local_rand_state;
        *d_world  = new hitable_list(d_list, num_objs);

        vec3 lookfrom(0,0,20);
        vec3 lookat(0,0,0.2);
        float dist_to_focus = 10.0; (lookfrom-lookat).length();
        float aperture = 0.1;
        *d_camera   = new camera(lookfrom,
                                 lookat,
                                 vec3(0,1,0),
                                 30.0,
                                 float(nx)/float(ny),
                                 aperture,
                                 dist_to_focus);
    }
}

// The goal of this world creation is to make a world that is completely load balanced.
__global__ void create_world_unbalanced(hitable **d_list, hitable **d_world, camera **d_camera, int nx, int ny, hiprandState *rand_state, int num_objs) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        int i = 0;
        int xtrans = -20;
        int ytrans = -20;
        int width = (int)sqrt((float)num_objs);
        float xspace = 1;
        float radius = xspace/2.0;

        int num_layers = 2;

        // MEtal layer.
        for(int a = -width*xspace; a < width*xspace; a+=xspace) {
            for(int b = -width/num_layers; b < width/num_layers; b+=xspace) {
                float choose_mat = RND;
                vec3 center(a + xtrans, b + ytrans, 0.2);
                d_list[i++] = new sphere(center, radius, new metal(vec3(0.7, 0.6, 0.5), 0.0));
            }
        }

        // Dielectric layer
        for(int a = -width*xspace; a < width*xspace; a+=xspace) {
            for(int b = -width/num_layers; b < width/num_layers; b+=xspace) {
                float choose_mat = RND;
                vec3 center(a + xtrans, b + ytrans, 0.2 + 2*radius);
                d_list[i++] = new sphere(center, xspace/2.0, new dielectric(1.5));
            }
        }
        
        *rand_state = local_rand_state;
        *d_world  = new hitable_list(d_list, num_objs);

        vec3 lookfrom(0,0,70);
        vec3 lookat(0,0,0.2);
        float dist_to_focus = 10.0; (lookfrom-lookat).length();
        float aperture = 0.1;
        *d_camera   = new camera(lookfrom,
                                 lookat,
                                 vec3(0,1,0),
                                 30.0,
                                 float(nx)/float(ny),
                                 aperture,
                                 dist_to_focus);
    }
}

// Moves the camera's origin to create a new scene
__global__ void move_cam(camera **d_camera, int steps = 1) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        (*d_camera)->origin += steps * vec3(0, 0, -0.1);
    }
}

__global__ void free_world(hitable **d_list, hitable **d_world, camera **d_camera, int num_objs) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        for (int i = 0; i < num_objs; i++) {
            delete ((sphere *)d_list[i])->mat_ptr;
            delete d_list[i];
        }
        delete *d_world;
        delete *d_camera;
    }
}

// Writes the image to a ppm file.
void write_frame_buffer(vec3 *frame_buffer, int nx, int ny, int max_x, int max_y) {
    FILE *f = fopen("output.ppm", "w");
    fprintf(f, "P3\n%d %d\n255\n", max_x, max_y);
    for(int j = 0; j < max_y; j++) {
        for(int i = 0; i < max_x; i++) {
            vec3 col = frame_buffer[j*max_x + i];
            int ir = int(255.99 * col[0]);
            int ig = int(255.99 * col[1]);
            int ib = int(255.99 * col[2]);
            fprintf(f, "%d %d %d ", ir, ig, ib);
        }
        fprintf(f, "\n");
    }
    fclose(f);
}

// Writes the image to a ppm file.
void write_frame_buffer_filename(vec3 *frame_buffer, int nx, int ny, int max_x, int max_y, std::string filename) {
    FILE *f = fopen(filename.c_str(), "w");
    fprintf(f, "P3\n%d %d\n255\n", max_x, max_y);
    for(int j = 0; j < max_y; j++) {
        for(int i = 0; i < max_x; i++) {
            vec3 col = frame_buffer[j*max_x + i];
            int ir = int(255.99 * col[0]);
            int ig = int(255.99 * col[1]);
            int ib = int(255.99 * col[2]);
            fprintf(f, "%d %d %d ", ir, ig, ib);
        }
        fprintf(f, "\n");
    }
    fclose(f);
}

// Renders a single image and writes it to a ppm file.
void test_render(int image_height, int image_width, int samples_per_pixel, int load_balancing_type) {
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << image_width << "x" << image_height << " image with " << samples_per_pixel << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = image_width*image_height;
    size_t frame_buffer_size = num_pixels*sizeof(vec3);

    // allocate frame_buffer
    vec3 *frame_buffer;
    checkCudaErrors(hipMallocManaged((void **)&frame_buffer, frame_buffer_size));

    // allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));
    hiprandState *d_rand_state2;
    checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1*sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init<<<1, 1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // make our world of hitables & the camera
    hitable **d_list;
    checkCudaErrors(hipMalloc((void **)&d_list, num_hitables*sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    if (load_balancing_type == NORMAL) {
        create_world<<<1, 1>>>(d_list, d_world, d_camera, image_width, image_height, d_rand_state2, num_hitables);
    } else if (load_balancing_type == UNBALANCED) {
        create_world_unbalanced<<<1, 1>>>(d_list, d_world, d_camera, image_width, image_height, d_rand_state2, num_hitables);
    } else if (load_balancing_type == BALANCED) {
        create_world_balanced<<<1, 1>>>(d_list, d_world, d_camera, image_width, image_height, d_rand_state2, num_hitables);
    }
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(image_width/tx+1,image_height/ty+1);
    dim3 threads(tx,ty);
    // render_init<<<blocks, threads>>>(image_width, image_height, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    // render<<<blocks, threads>>>(frame_buffer, image_width, image_height, samples_per_pixel, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output frame_buffer as Image
    write_frame_buffer(frame_buffer, image_width, image_height, image_width, image_height);

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world,d_camera, num_hitables);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(frame_buffer));

    hipDeviceReset();
}

void benchmark_single(int image_height, int image_width, int samples_per_pixel, int num_frames_to_render, int load_balancing_type, int straggler) {
    int tx = 8;
    int ty = 8;

    std::cerr << "Benchmarking the rendering of " << image_width << "x" << image_height << " images with " << samples_per_pixel << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = image_width*image_height;
    size_t frame_buffer_size = num_pixels*sizeof(vec3);

    // allocate frame_buffer
    vec3 *frame_buffer;
    checkCudaErrors(hipMallocManaged((void **)&frame_buffer, frame_buffer_size));

    // allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));
    hiprandState *d_rand_state2;
    checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1*sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init<<<1,1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // make our world of hitables & the camera
    hitable **d_list;
    checkCudaErrors(hipMalloc((void **)&d_list, num_hitables * sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    if (load_balancing_type == NORMAL) {
        create_world<<<1, 1>>>(d_list, d_world, d_camera, image_width, image_height, d_rand_state2, num_hitables);
    } else if (load_balancing_type == UNBALANCED) {
        create_world_unbalanced<<<1, 1>>>(d_list, d_world, d_camera, image_width, image_height, d_rand_state2, num_hitables);
    } else if (load_balancing_type == BALANCED) {
        create_world_balanced<<<1, 1>>>(d_list, d_world, d_camera, image_width, image_height, d_rand_state2, num_hitables);
    }
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(image_width/tx+1,image_height/ty+1);
    dim3 threads(tx,ty);
    render_init_tiled<<<blocks, threads>>>(image_width, image_height, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    for (int i = 0; i < num_frames_to_render; i++) {
        // Render the current frame and make sure it worked.
        render_tiled<<<blocks, threads>>>(frame_buffer, image_width, image_height, 0, 0, image_width, image_height, samples_per_pixel, d_camera, d_world, d_rand_state);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());
        // Move the camera to create the next frame.
        move_cam<<<blocks, threads>>>(d_camera);
    }
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds to generate " << num_frames_to_render << " frames.\n";
    std::cerr << "Average FPS: " << (double)num_frames_to_render / timer_seconds << "\n";

    // Output frame_buffer as Image
    write_frame_buffer(frame_buffer, image_width, image_height, image_width, image_height);

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world,d_camera, num_hitables);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(frame_buffer));

    hipDeviceReset();
}

void benchmark_tiled(int image_height, int image_width, int samples_per_pixel, int num_frames_to_render, int load_balancing_type, int num_gpus = -1, int straggler = 0) {
    int tx = 8;
    int ty = 8;

    std::cerr << "Benchmarking the rendering of " << image_width << "x" << image_height << " images with " << samples_per_pixel << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = image_width * image_height;

    // allocate frame_buffer
    vec3 *frame_buffer;
    checkCudaErrors(hipMallocManaged((void **)&frame_buffer, num_pixels * sizeof(vec3)));

    int available_gpus;
    checkCudaErrors(hipGetDeviceCount(&available_gpus));
    if (num_gpus > available_gpus) {
        std::cerr << "requeted more than available GPUs, capping." << std::endl;
        num_gpus = available_gpus;
    } else if (num_gpus == -1) {
        num_gpus = available_gpus;
    }

    int num_streams = num_gpus;
    
    int per_gpu_width = image_width;
    int per_gpu_height = image_height / num_gpus;
    int num_pixels_per_gpu = per_gpu_width * per_gpu_height;

    dim3 blocks(per_gpu_width / tx + 1, per_gpu_height / ty + 1);
    dim3 threads(tx, ty);

    // allocate random state
    using std::vector;
    vector<hiprandState *> d_rand_state(num_gpus);
    vector<hiprandState *> d_rand_state2(num_gpus);
    vector<hipStream_t> streams(num_streams);
    vector<hitable **> d_list(num_gpus);
    vector<hitable **> d_world(num_gpus);
    vector<camera **> d_camera(num_gpus);
    for (int gpu_id = 0; gpu_id < num_gpus; gpu_id++) {
        checkCudaErrors(hipSetDevice(gpu_id));
        checkCudaErrors(hipMalloc((void **) &d_rand_state[gpu_id], num_pixels_per_gpu * sizeof(hiprandState)));
        checkCudaErrors(hipMalloc((void **) &d_rand_state2[gpu_id], 1 * sizeof(hiprandState)));
        checkCudaErrors(hipStreamCreate(&streams[gpu_id]));
        rand_init<<<1, 1>>>(d_rand_state2[gpu_id]);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipMalloc((void **) &d_list[gpu_id], num_hitables * sizeof(hitable *)));
        checkCudaErrors(hipMalloc((void **) &d_world[gpu_id], 1 * sizeof(hitable *)));
        checkCudaErrors(hipMalloc((void **) &d_camera[gpu_id], 1 * sizeof(camera *)));
        hipDeviceSynchronize();
        if (load_balancing_type == NORMAL) {
            create_world<<<1, 1>>>(d_list[gpu_id], d_world[gpu_id], d_camera[gpu_id], image_width, image_height, d_rand_state2[gpu_id], num_hitables);
        } else if (load_balancing_type == UNBALANCED) {
            create_world_unbalanced<<<1, 1>>>(d_list[gpu_id], d_world[gpu_id], d_camera[gpu_id], image_width, image_height, d_rand_state2[gpu_id], num_hitables);
        } else if (load_balancing_type == BALANCED) {
            create_world_balanced<<<1, 1>>>(d_list[gpu_id], d_world[gpu_id], d_camera[gpu_id], image_width, image_height, d_rand_state2[gpu_id], num_hitables);
        }
        checkCudaErrors(hipGetLastError());
    }

    clock_t start, stop;
    start = clock();
    // Render our buffer
    for (int gpu_id = 0; gpu_id < num_gpus; gpu_id++) {
        checkCudaErrors(hipSetDevice(gpu_id));
        render_init_tiled<<<blocks, threads, 0, streams[gpu_id]>>>(per_gpu_width, per_gpu_height, d_rand_state[gpu_id]);
        // render_init<<<blocks, threads, 0, streams[gpu_id]>>>(image_width, image_height, d_rand_state[gpu_id], gpu_id, num_gpus);
    }
    for (int stream_id = 0; stream_id < num_streams; stream_id++) {
        checkCudaErrors(hipStreamSynchronize(streams[stream_id]));
    }
    for (int i = 0; i < num_frames_to_render; i++) {
        for (int gpu_id = 0; gpu_id < num_gpus; gpu_id++) {
            checkCudaErrors(hipSetDevice(gpu_id));
            int x_start = 0, y_start = image_height * gpu_id / num_gpus;
            int make_straggler = 0;
            if (straggler == 1 && num_gpus > 1 && gpu_id == 1) {
                make_straggler = 1;
            }
            render_tiled<<<blocks, threads, 0, streams[gpu_id]>>>(frame_buffer, image_width, image_height, x_start, y_start, per_gpu_width, per_gpu_height,
                samples_per_pixel, d_camera[gpu_id], d_world[gpu_id], d_rand_state[gpu_id], make_straggler);
            checkCudaErrors(hipGetLastError());
            move_cam<<<1, 1, 0, streams[gpu_id]>>>(d_camera[gpu_id]);
            checkCudaErrors(hipGetLastError());
        }
        for (int gpu_id = 0; gpu_id < num_gpus; gpu_id++) {
            checkCudaErrors(hipStreamSynchronize(streams[gpu_id]));
        }
    }

    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds to generate " << num_frames_to_render << " frames.\n";
    std::cerr << "Average FPS: " << (double)num_frames_to_render / timer_seconds << "\n";

    // Output frame_buffer as Image
    write_frame_buffer(frame_buffer, image_width, image_height, image_width, image_height);

    // clean up
    for (int gpu_id = 0; gpu_id < num_gpus; gpu_id++) {
        checkCudaErrors(hipSetDevice(gpu_id));
        free_world<<<1, 1>>>(d_list[gpu_id], d_world[gpu_id], d_camera[gpu_id], num_hitables);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipFree(d_camera[gpu_id]));
        checkCudaErrors(hipFree(d_world[gpu_id]));
        checkCudaErrors(hipFree(d_list[gpu_id]));
        checkCudaErrors(hipStreamDestroy(streams[gpu_id]));
        checkCudaErrors(hipFree(d_rand_state2[gpu_id]));
        checkCudaErrors(hipFree(d_rand_state[gpu_id]));
    }
    checkCudaErrors(hipFree(frame_buffer));

    hipDeviceReset();
}

__global__ void update_camera_origin(vec3 new_origin, camera **d_camera) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        (*d_camera)->origin.e[0] = new_origin.x();
        (*d_camera)->origin.e[1] = new_origin.y();
        (*d_camera)->origin.e[2] = new_origin.z();
    }
}

void init_camera_origins_for_frames(vec3 *camera_origins_for_frames, int num_frames_to_render, vec3& current_origin) {
    auto move_vector = vec3(0, 0, -0.1);
    camera_origins_for_frames[0] = vec3(current_origin);
    for (int i = 1; i < num_frames_to_render; i++) {
        // Each subsequent frame is offset by the move vector.
        camera_origins_for_frames[i].e[0] = camera_origins_for_frames[i-1].x();
        camera_origins_for_frames[i].e[1] = camera_origins_for_frames[i-1].y();
        camera_origins_for_frames[i].e[2] = camera_origins_for_frames[i-1].z();
    }
}

enum message_tag {
    KILL_SIGNAL,
    CAMERA_ORIGIN_INFO,
    FRAME_BUFFER_BACK,
    STOP_WORK_SIGNAL,
};

void benchmark_frame(int argc, char **argv, int image_height, int image_width, int samples_per_pixel, int num_frames_to_render, int load_balancing_type, int straggler) {
    int num_procs, rank;

    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &num_procs);
    
    if (num_procs < 2) {
        std::cerr << "!!!!!!!!Hey you, you only used 1 process. When you run framed we use 1 mpi task per gpu and an extra mpi task to dispatch work. So if you want to run with 1 GPU use -N 2, if you want to run with 2 GPUs use -N 3 etc.\n";
        exit(1);
    }

    if (rank == 0) {
        std::cerr << "Benchmarking the rendering of " << num_frames_to_render << " " << image_width << "x" << image_height << " images with " << samples_per_pixel << " samples per pixel " << std::endl;
        std::cerr << "num_gpus: " << num_procs - 1 << std::endl;
    }

    MPI_Datatype MPI_Vec3;
    int nitems = 3;
    int blocklengths[3] = { 1, 1, 1 };
    MPI_Aint offsets[3];
    offsets[0] = offsetof(vec3, e[0]);
    offsets[1] = offsetof(vec3, e[1]);
    offsets[2] = offsetof(vec3, e[2]);
    MPI_Datatype types[3] = { MPI_FLOAT, MPI_FLOAT, MPI_FLOAT };
    MPI_Type_create_struct(nitems, blocklengths, offsets, types, &MPI_Vec3);
    MPI_Type_commit(&MPI_Vec3);

    if (rank > 0) {
        checkCudaErrors(hipSetDevice(rank - 1));
    } else {
        checkCudaErrors(hipSetDevice(0));
    }

    int num_pixels = image_width * image_height;

    // every machine init random and world
    hiprandState *d_rand_state;
    hiprandState *d_rand_state2;
    hitable **d_list;
    hitable **d_world;
    camera **d_camera;
    if (rank > 0) {
        checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels * sizeof(hiprandState)));
        checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1*sizeof(hiprandState)));
        rand_init<<<1, 1>>>(d_rand_state2);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipMalloc((void **)&d_list, num_hitables * sizeof(hitable *)));
        checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
        checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
        if (load_balancing_type == NORMAL) {
            create_world<<<1, 1>>>(d_list, d_world, d_camera, image_width, image_height, d_rand_state2, num_hitables);
        } else if (load_balancing_type == UNBALANCED) {
            create_world_unbalanced<<<1, 1>>>(d_list, d_world, d_camera, image_width, image_height, d_rand_state2, num_hitables);
        } else if (load_balancing_type == BALANCED) {
            create_world_balanced<<<1, 1>>>(d_list, d_world, d_camera, image_width, image_height, d_rand_state2, num_hitables);
        }
        checkCudaErrors(hipGetLastError());
    }

    vec3 *frame_buffer;
    checkCudaErrors(hipMallocManaged((void **)&frame_buffer, num_pixels * sizeof(vec3)));

    vec3 *camera_origins_for_frames;
    if (rank == 0) {
        //checkCudaErrors(hipMallocManaged((void **)&camera_origins_for_frames, num_frames_to_render * sizeof(vec3)));
        camera_origins_for_frames = new vec3[num_frames_to_render];
        if (load_balancing_type == NORMAL) {
            auto origin = vec3(13, 2, 3);
            init_camera_origins_for_frames(camera_origins_for_frames, num_frames_to_render, origin);
        } else if (load_balancing_type == UNBALANCED) {
            auto origin = vec3(0,0,70);
        } else if (load_balancing_type == BALANCED) {
            auto origin = vec3(0,0,20);
        }
    }

    clock_t start, stop;
    if (rank == 0) {
        start = clock();
    }

    int tx = 8;
    int ty = 8;

    dim3 blocks(image_width/tx + 1, image_height/ty + 1);
    dim3 threads(tx, ty);
    if (rank > 0) {
        render_init_tiled<<<blocks, threads>>>(image_width, image_height, d_rand_state);
    }
    checkCudaErrors(hipGetLastError());

    if (rank == 0) {
        std::vector<int> free_gpus;
        std::vector<int> remaining_frames;
        for (int i = 1; i < num_procs; i++) {
            free_gpus.push_back(i);
        }
        for (int i = 0; i < num_frames_to_render; i++) {
            remaining_frames.push_back(i);
        }

        std::map<int, int> work_assignment; // gpu_id -> frame_id, ordered on gpu_id.
        // only those in progress are in the work_assignment map.
        std::map<int, clock_t> work_assignment_time;
        // std::unordered_map<int, MPI_Request> work_requests; // gpu_id -> Request
        MPI_Request work_requests[num_procs-1];
        for (int i = 0; i < num_procs-1; i++) {
            work_requests[i] = MPI_REQUEST_NULL;
        }
        MPI_Status status;
        int index;


        vec3 **recv_frame_buffer = new vec3 *[num_procs];
        for (int i = 1; i < num_procs; i++) {
            recv_frame_buffer[i] = new vec3[num_pixels];
        }

        MPI_Request ignore;

        while (true) {
            if (remaining_frames.size() == 0 && free_gpus.size() == num_procs - 1) {
                // no work to be done and no work to be waited so we kill everyone and get out
                int signal = 267;
                for (int other = 1; other < num_procs; other++) {
                    MPI_Isend(&signal, 1, MPI_INT, other, KILL_SIGNAL, MPI_COMM_WORLD, &ignore);
                }
                for (int i = 1; i < num_procs; i++) {
                    delete[] recv_frame_buffer[i];
                }
                delete[] recv_frame_buffer;
                break;
            }

            if (free_gpus.size() > 0) {
                for(int gpu_id = 0; gpu_id < num_procs - 1; gpu_id++) {
                    if (work_assignment_time.find(gpu_id) != work_assignment_time.end() // gpu is assigned something
                        && clock() - work_assignment_time[gpu_id] > 1.29 * pow(10, 9) * 2) { // over 2s means straggler
                        int frame_id = work_assignment[gpu_id];
                        remaining_frames.insert(remaining_frames.begin(), frame_id); // put frame at front of queue
                        work_assignment_time.erase(gpu_id); // dont look at this straggler gpu again
                        break;
                    }
                }
            }

            if (remaining_frames.size() == 0 || free_gpus.size() == 0) {
                MPI_Waitany(num_procs-1, work_requests, &index, &status);
                int gpu_id = index + 1;
                free_gpus.push_back(gpu_id);
                // Remove the gpu from the frame to gpu map.
                int frame_id = work_assignment[gpu_id];
                work_assignment.erase(gpu_id);
                if  (work_assignment_time.find(gpu_id) != work_assignment_time.end())
                    work_assignment_time.erase(gpu_id);
                work_requests[gpu_id - 1] = MPI_REQUEST_NULL;
                if (remaining_frames.size() == 0) {
                    continue;
                }
            }

            auto frame_iter = remaining_frames.begin();
            int frame = *frame_iter;
            remaining_frames.erase(frame_iter);
            auto gpu_it = free_gpus.begin();
            int gpu = *gpu_it;
            free_gpus.erase(gpu_it);
            work_assignment[gpu] = frame;
            work_assignment_time[gpu] = clock();
            // pop a frame
            // pop a gpu
            vec3 camera_origin = camera_origins_for_frames[frame];
            MPI_Isend(camera_origin.e, 1, MPI_Vec3, gpu, CAMERA_ORIGIN_INFO, MPI_COMM_WORLD, &ignore);
            // send camera to remote gpu
            MPI_Irecv(recv_frame_buffer[gpu], num_pixels, MPI_Vec3, gpu, FRAME_BUFFER_BACK, MPI_COMM_WORLD, &work_requests[gpu-1]);
            // recv into corresponding frame_buffer
        }

    } else {
        vec3 camera_origin;
        int signal;
        MPI_Request requests[2];
        MPI_Irecv(&signal, 1, MPI_INT, 0, KILL_SIGNAL, MPI_COMM_WORLD, &requests[0]);
        int which_request;
        MPI_Status status;
        while (true) {
            MPI_Irecv(camera_origin.e, 1, MPI_Vec3, 0, CAMERA_ORIGIN_INFO, MPI_COMM_WORLD, &requests[1]);
            MPI_Waitany(2, requests, &which_request, &status);
            if (which_request == 0) {
                // signal received
                if (signal == 267) {
                    break;
                }
            }
            // get camera origin info
            update_camera_origin<<<1, 1>>>(camera_origin, d_camera);
            // update d_camera
            int make_straggler = 0;
            if (straggler == 1 && rank == 1 && num_procs > 2) {
                make_straggler = 1;
            }
            render_tiled<<<blocks, threads>>>(frame_buffer, image_width, image_height, 0, 0, image_width, image_height,
                samples_per_pixel, d_camera, d_world, d_rand_state, make_straggler);
            // render
            // send back frame
            checkCudaErrors(hipDeviceSynchronize());
            // if (straggler == 1) {
            //     if (rand() % 2 == 0)
            //         sleep(5); // sleep 5 secs
            // }
            MPI_Send(frame_buffer, num_pixels, MPI_Vec3, 0, FRAME_BUFFER_BACK, MPI_COMM_WORLD);
        }
    }

    if (rank == 0) {
        stop = clock();
        double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
        std::cerr << "took " << timer_seconds << " seconds to generate " << num_frames_to_render << " frames.\n";
        std::cerr << "Average FPS: " << (double)num_frames_to_render / timer_seconds << "\n";
    }

    // if (rank == 0) {
    //     checkCudaErrors(hipFree(camera_origins_for_frames));
    // }
    if (rank > 0) {
        checkCudaErrors(hipFree(frame_buffer));
        free_world<<<1, 1>>>(d_list, d_world, d_camera, num_hitables);
        checkCudaErrors(hipFree(d_camera));
        checkCudaErrors(hipFree(d_world));
        checkCudaErrors(hipFree(d_list));
        checkCudaErrors(hipFree(d_rand_state2));
        checkCudaErrors(hipFree(d_rand_state));
    }

    MPI_Finalize();

}

// Benchmarks the throughput of a rendering type.
void benchmark_rendering(int argc, char **argv, std::string rendering_strategy, int image_height, int image_width, int samples_per_pixel, int num_frames_to_render, int requested_gpus, int load_balancing_type, int straggler) {
    if (strcmp(rendering_strategy.c_str(), "singlenode") == 0) {
        benchmark_single(image_height, image_width, samples_per_pixel, num_frames_to_render, load_balancing_type, straggler);
    } else if (strcmp(rendering_strategy.c_str(), "tiled") == 0) {
        benchmark_tiled(image_height, image_width, samples_per_pixel, num_frames_to_render, load_balancing_type, requested_gpus, straggler);
    } else if (strcmp(rendering_strategy.c_str(), "frame") == 0) {
        benchmark_frame(argc, argv, image_height, image_width, samples_per_pixel, num_frames_to_render, load_balancing_type, straggler);
    }
}

int main(int argc, char **argv) {
    // Parse Args
    if (find_arg_idx(argc, argv, "-h") >= 0) {
        std::cout << "Options:" << std::endl;
        std::cout << "-h: see this help" << std::endl;
        std::cout << "-t <int>: type, 0 = test, 1 = benchmark" << std::endl;
        std::cout << "-r <rendering strategy>: singlenode/tiled/frame" << std::endl;
        std::cout << "-v <int>: vertical height of image in pixels" << std::endl;
        std::cout << "-w <int>: width of image in pixels" << std::endl;
        std::cout << "-s <int>: number of samples per pixel" << std::endl;
        std::cout << "-f <int>: number of frames to render" << std::endl;
        std::cout << "-g <int>: number of gpus to use" << std::endl;
        std::cout << "-l <int>: load balancing type, 0 = normal scene, 1 = unbalanced, 2 = perfectly balanced as all things should be" << std::endl;
        std::cout << "-z <int>: 0 = no straggler, 1 = one straggler" << std::endl;
        return 0;
    }

    int image_height = find_int_arg(argc, argv, "-v", 800);
    int image_width = find_int_arg(argc, argv, "-w", 1200);
    int samples_per_pixel = find_int_arg(argc, argv, "-s", 10);

    int load_balancing_type = find_int_arg(argc, argv, "-l", 0);
    if (load_balancing_type == 0) {
        num_hitables = (22*22 + 1 + 3);
    } else {
        num_hitables = 22*22;
    }

    int type = find_int_arg(argc, argv, "-t", 0);
    if (type == 0) {
        test_render(image_height, image_width, samples_per_pixel, load_balancing_type);
        return 0;
    }

    int num_frames_to_render = find_int_arg(argc, argv, "-f", 30);
    int requested_gpus = find_int_arg(argc, argv, "-g", -1);
    int straggler = find_int_arg(argc, argv, "-z", 0);

    std::string rendering_strategy = find_string_option(argc, argv, "-r", std::string("singlenode"));
    if (strcmp(rendering_strategy.c_str(), "singlenode") != 0  && strcmp(rendering_strategy.c_str(), "tiled") != 0 && strcmp(rendering_strategy.c_str(), "frame") != 0) {
        std::cerr << "Unknown rendering strategy: " << rendering_strategy << std::endl;
        return 1;
    }


    benchmark_rendering(argc, argv, rendering_strategy, image_height, image_width, samples_per_pixel, num_frames_to_render, requested_gpus, load_balancing_type, straggler);
}