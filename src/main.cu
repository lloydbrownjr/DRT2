#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <time.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>
#include "../common/vec3.h"
#include "../common/ray.h"
#include "sphere.h"
#include "hitable_list.h"
#include "camera.h"
#include "material.h"
#include "cuda_errors.h"
#include "../common/options.h"
#include "hip/hip_runtime.h"
// #include "nccl.h"
#include <omp.h>

// #define NCCLCHECK(cmd) do {                         \
//   ncclResult_t r = cmd;                             \
//   if (r!= ncclSuccess) {                            \
//     printf("Failed, NCCL error %s:%d '%s'\n",             \
//         __FILE__,__LINE__,ncclGetErrorString(r));   \
//     exit(EXIT_FAILURE);                             \
//   }                                                 \
// } while(0)

// This method determines the color of a ray going through the scene by tracing it through the scene and hitting objects.
// It has been modified to use CUDA as described below.
// Matching the C++ code would recurse enough into color() calls that
// it was blowing up the stack, so we have to turn this into a
// limited-depth loop instead.  Later code in the book limits to a max
// depth of 50, so we adapt this a few chapters early on the GPU.
__device__ vec3 color(const ray& r, hitable **world, hiprandState *local_rand_state) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0,1.0,1.0);
    for(int i = 0; i < 50; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            if(rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return vec3(0.0,0.0,0.0);
            }
        }
        else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f*(unit_direction.y() + 1.0f);
            vec3 c = (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return vec3(0.0,0.0,0.0); // exceeded recursion
}

__global__ void rand_init(hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state, int init_x, int init_y) {
    int i_local = threadIdx.x + blockIdx.x * blockDim.x;
    int j_local = threadIdx.y + blockIdx.y * blockDim.y;
    int i = i_local + init_x;
    int j = j_local + init_y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    int pixel_index_local = j_local*max_x + i_local;
    // Original: Each thread gets same seed, a different sequence number, no offset
    // hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
    // BUGFIX, see Issue#2: Each thread gets different seed, same sequence for
    // performance improvement of about 2x!
    hiprand_init(1984+pixel_index, 0, 0, &rand_state[pixel_index_local]);
}

__global__ void render(vec3 *frame_buffer, int max_x, int max_y, int ns, camera **cam, hitable **world, hiprandState *rand_state, int init_x, int init_y) {
    int i_local = threadIdx.x + blockIdx.x * blockDim.x + init_x;
    int j_local = threadIdx.y + blockIdx.y * blockDim.y + init_y;
    int i = i_local + init_x;
    int j = j_local + init_y;
    if((i >= max_x) || (j >= max_y)) return;
    // int pixel_index = j*max_x + i;
    int pixel_index_local = j_local*max_x + i_local;
    hiprandState local_rand_state = rand_state[pixel_index_local];
    vec3 col(0,0,0);
    for(int s=0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += color(r, world, &local_rand_state);
    }
    rand_state[pixel_index_local] = local_rand_state;
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    frame_buffer[pixel_index_local] = col;
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(hitable **d_list, hitable **d_world, camera **d_camera, int nx, int ny, hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        d_list[0] = new sphere(vec3(0,-1000.0,-1), 1000,
                               new lambertian(vec3(0.5, 0.5, 0.5)));
        int i = 1;
        for(int a = -11; a < 11; a++) {
            for(int b = -11; b < 11; b++) {
                float choose_mat = RND;
                vec3 center(a+RND,0.2,b+RND);
                if(choose_mat < 0.8f) {
                    d_list[i++] = new sphere(center, 0.2,
                                             new lambertian(vec3(RND*RND, RND*RND, RND*RND)));
                }
                else if(choose_mat < 0.95f) {
                    d_list[i++] = new sphere(center, 0.2,
                                             new metal(vec3(0.5f*(1.0f+RND), 0.5f*(1.0f+RND), 0.5f*(1.0f+RND)), 0.5f*RND));
                }
                else {
                    d_list[i++] = new sphere(center, 0.2, new dielectric(1.5));
                }
            }
        }
        d_list[i++] = new sphere(vec3(0, 1,0),  1.0, new dielectric(1.5));
        d_list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
        d_list[i++] = new sphere(vec3(4, 1, 0),  1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));
        *rand_state = local_rand_state;
        *d_world  = new hitable_list(d_list, 22*22+1+3);

        vec3 lookfrom(13,2,3);
        vec3 lookat(0,0,0);
        float dist_to_focus = 10.0; (lookfrom-lookat).length();
        float aperture = 0.1;
        *d_camera   = new camera(lookfrom,
                                 lookat,
                                 vec3(0,1,0),
                                 30.0,
                                 float(nx)/float(ny),
                                 aperture,
                                 dist_to_focus);
    }
}

// Moves the camera's origin to create a new scene
__global__ void move_cam(camera **d_camera) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        (*d_camera)->origin += vec3(0,0,-0.1);
    }
}

__global__ void free_world(hitable **d_list, hitable **d_world, camera **d_camera) {
    for(int i=0; i < 22*22+1+3; i++) {
        delete ((sphere *)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete *d_world;
    delete *d_camera;
}

// Writes the image to a ppm file.
void write_frame_buffer(vec3 *frame_buffer, int nx, int ny, int max_x, int max_y) {
    FILE *f = fopen("output.ppm", "w");
    fprintf(f, "P3\n%d %d\n255\n", max_x, max_y);
    for(int j=0; j < max_y; j++) {
        for(int i=0; i < max_x; i++) {
            vec3 col = frame_buffer[j*max_x + i];
            int ir = int(255.99*col[0]);
            int ig = int(255.99*col[1]);
            int ib = int(255.99*col[2]);
            fprintf(f, "%d %d %d ", ir, ig, ib);
        }
        fprintf(f, "\n");
    }
    fclose(f);
}

// Writes the image to a ppm file.
void write_frame_buffer_arr(vec3 **frame_buffer_arr, int nx, int ny, int max_x, int max_y) {
    int size = sizeof frame_buffer_arr / sizeof frame_buffer_arr[0];
    int slice_height = max_y/size;
    FILE *f = fopen("output.ppm", "w");
    fprintf(f, "P3\n%d %d\n255\n", max_x, max_y);
    for(int j=0; j < max_y; j++) {
        for(int i=0; i < max_x; i++) {
            int slice = j/slice_height;
            vec3 col = frame_buffer_arr[slice][(j-(slice*slice_height))*max_x + i];
            int ir = int(255.99*col[0]);
            int ig = int(255.99*col[1]);
            int ib = int(255.99*col[2]);
            fprintf(f, "%d %d %d ", ir, ig, ib);
        }
        fprintf(f, "\n");
    }
    fclose(f);
}

// Renders a single image and writes it to a ppm file.
void test_render(int image_height, int image_width, int samples_per_pixel) {
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << image_width << "x" << image_height << " image with " << samples_per_pixel << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = image_width*image_height;
    size_t frame_buffer_size = num_pixels*sizeof(vec3);

    // allocate frame_buffer
    vec3 *frame_buffer;
    checkCudaErrors(hipMallocManaged((void **)&frame_buffer, frame_buffer_size));

    // allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));
    hiprandState *d_rand_state2;
    checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1*sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init<<<1,1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // make our world of hitables & the camera
    hitable **d_list;
    int num_hitables = 22*22+1+3;
    checkCudaErrors(hipMalloc((void **)&d_list, num_hitables*sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    create_world<<<1,1>>>(d_list, d_world, d_camera, image_width, image_height, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(image_width/tx+1,image_height/ty+1);
    dim3 threads(tx,ty);
    render_init<<<blocks, threads>>>(image_width, image_height, d_rand_state, 0, 0);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(frame_buffer, image_width, image_height, samples_per_pixel, d_camera, d_world, d_rand_state, 0, 0);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output frame_buffer as Image
    write_frame_buffer(frame_buffer, image_width, image_height, image_width, image_height);

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world,d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(frame_buffer));

    hipDeviceReset();
}

void benchmark_single(int image_height, int image_width, int samples_per_pixel, int num_frames_to_render) {
    int tx = 8;
    int ty = 8;

    std::cerr << "Benchmarking the rendering of " << image_width << "x" << image_height << " images with " << samples_per_pixel << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = image_width*image_height;
    size_t frame_buffer_size = num_pixels*sizeof(vec3);

    // allocate frame_buffer
    vec3 *frame_buffer;
    checkCudaErrors(hipMallocManaged((void **)&frame_buffer, frame_buffer_size));

    // allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));
    hiprandState *d_rand_state2;
    checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1*sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init<<<1,1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // make our world of hitables & the camera
    hitable **d_list;
    int num_hitables = 22*22+1+3;
    checkCudaErrors(hipMalloc((void **)&d_list, num_hitables*sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    create_world<<<1,1>>>(d_list, d_world, d_camera, image_width, image_height, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(image_width/tx+1,image_height/ty+1);
    dim3 threads(tx,ty);
    render_init<<<blocks, threads>>>(image_width, image_height, d_rand_state, 0, 0);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    for (int i = 0; i < num_frames_to_render; i++) {
        // Render the current frame and make sure it worked.
        render<<<blocks, threads>>>(frame_buffer, image_width, image_height, samples_per_pixel, d_camera, d_world, d_rand_state, 0, 0);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());
        // Move the camera to create the next frame.
        move_cam<<<blocks, threads>>>(d_camera);
    }
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds to generate " << num_frames_to_render << " frames.\n";
    std::cerr << "Average FPS: " << (double)num_frames_to_render / timer_seconds << "\n";

    // Output frame_buffer as Image
    write_frame_buffer(frame_buffer, image_width, image_height, image_width, image_height);

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world,d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(frame_buffer));

    hipDeviceReset();
}

void benchmark_tiled(int image_height, int image_width, int samples_per_pixel, int num_frames_to_render, int num_devices = 4) {
    int tx = 8;
    int ty = 8;

    std::cerr << "Benchmarking the rendering of " << image_width << "x" << image_height << " images with " << samples_per_pixel << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    // ncclComm_t comms[num_devices];

    //managing X devices
    int nDev = num_devices;
    // int devs[num_devices] = {};
    // for (int i = 0; i < num_devices; i++)
    //     devs[i] = i;

    //allocating and initializing device buffers
    hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*nDev);

    int num_pixels = image_width*image_height;
    size_t frame_buffer_size = num_pixels*sizeof(vec3);

    int image_width_dev = image_width;
    int image_height_dev = image_height / nDev;
    if (image_width_dev * image_height_dev * nDev != image_height * image_width) {
        std::cerr << "Not an even split." << std::endl;
        exit(1);
    }
    int num_pixels_dev = num_pixels / nDev;
    size_t frame_buffer_size_dev = frame_buffer_size / nDev;

    vec3** frame_buffer = (vec3**)malloc(nDev * sizeof(vec3*));
    hiprandState** d_rand_state = (hiprandState**)malloc(nDev * sizeof(hiprandState*));
    hiprandState** d_rand_state2 = (hiprandState**)malloc(nDev * sizeof(hiprandState*));
    hitable ***d_list = (hitable***)malloc(nDev * sizeof(hitable**));
    int num_hitables = 22*22+1+3;
    hitable ***d_world = (hitable***)malloc(nDev * sizeof(hitable**));
    camera ***d_camera = (camera***)malloc(nDev * sizeof(camera**));

    for (int i = 0; i < nDev; ++i) {
        checkCudaErrors(hipSetDevice(i));

        // allocate frame_buffer
        checkCudaErrors(hipMallocManaged((void **)&frame_buffer[i], frame_buffer_size_dev));

        // allocate random state
        checkCudaErrors(hipMalloc((void **)&d_rand_state[i], num_pixels_dev*sizeof(hiprandState)));
        checkCudaErrors(hipMalloc((void **)&d_rand_state2[i], 1*sizeof(hiprandState)));

        // we need that 2nd random state to be initialized for the world creation
        rand_init<<<1,1>>>(d_rand_state2[i]);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

        // make our world of hitables & the camera
        checkCudaErrors(hipMalloc((void **)&d_list[i], num_hitables*sizeof(hitable *)));
        checkCudaErrors(hipMalloc((void **)&d_world[i], sizeof(hitable *)));
        checkCudaErrors(hipMalloc((void **)&d_camera[i], sizeof(camera *)));
        create_world<<<1,1>>>(d_list[i], d_world[i], d_camera[i], image_width_dev, image_height_dev, d_rand_state2[i]);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

        checkCudaErrors(hipStreamCreate(s+i));
    }

    //initializing NCCL
    // NCCLCHECK(ncclCommInitAll(comms, nDev, devs));

    clock_t start, stop;
    start = clock();

    dim3 blocks(image_width_dev/tx+1,image_height_dev/ty+1);
    dim3 threads(tx,ty);

    #pragma omp parallel for
    for (int i = 0; i < nDev; ++i) {
        checkCudaErrors(hipSetDevice(i));
        // Render our buffer
        render_init<<<blocks, threads>>>(image_width_dev, image_height_dev, d_rand_state[i], 0, i*image_height_dev);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

        for (int f = 0; f < num_frames_to_render; f++) {
            // Render the current frame and make sure it worked.
            render<<<blocks, threads>>>(frame_buffer[i], image_width_dev, image_height_dev, samples_per_pixel, d_camera[i], d_world[i], d_rand_state[i], 0, i*image_height_dev);
            checkCudaErrors(hipGetLastError());
            checkCudaErrors(hipDeviceSynchronize());

            checkCudaErrors(hipSetDevice(i));
            // Move the camera to create the next frame.
            move_cam<<<blocks, threads>>>(d_camera[i]);
        }
    }

    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds to generate " << num_frames_to_render << " frames.\n";
    std::cerr << "Average FPS: " << (double)num_frames_to_render / timer_seconds << "\n";

    // Output frame_buffer as Image
    write_frame_buffer_arr(frame_buffer, image_width, image_height, image_width, image_height);

    //free device buffers
    for (int i = 0; i < nDev; ++i) {
        checkCudaErrors(hipSetDevice(i));
        // clean up
        checkCudaErrors(hipDeviceSynchronize());
        free_world<<<1,1>>>(d_list[i],d_world[i],d_camera[i]);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipFree(d_camera[i]));
        checkCudaErrors(hipFree(d_world[i]));
        checkCudaErrors(hipFree(d_list[i]));
        checkCudaErrors(hipFree(d_rand_state[i]));
        checkCudaErrors(hipFree(d_rand_state2[i]));
        checkCudaErrors(hipFree(frame_buffer[i]));
    }

    //finalizing NCCL
    // for(int i = 0; i < nDev; ++i)
    //     ncclCommDestroy(comms[i]);

    printf("Success \n");

    hipDeviceReset();
}

void benchmark_frame(int image_height, int image_width, int samples_per_pixel, int num_frames_to_render) {
    std::cerr << "Not implemented." << std::endl;
    exit(1);
}

// Benchmarks the throughput of a rendering type.
void benchmark_rendering(std::string rendering_strategy, int image_height, int image_width, int samples_per_pixel, int num_frames_to_render, int num_devices = 4) {
    if (strcmp(rendering_strategy.c_str(), "singlenode") == 0) {
        benchmark_single(image_height, image_width, samples_per_pixel, num_frames_to_render);
    } else if (strcmp(rendering_strategy.c_str(), "tiled") == 0) {
        benchmark_tiled(image_height, image_width, samples_per_pixel, num_frames_to_render, num_devices);
    } else if (strcmp(rendering_strategy.c_str(), "frame") == 0) {
        benchmark_frame(image_height, image_width, samples_per_pixel, num_frames_to_render);
    }
}

int main(int argc, char **argv) {
    // Parse Args
    if (find_arg_idx(argc, argv, "-h") >= 0) {
        std::cout << "Options:" << std::endl;
        std::cout << "-h: see this help" << std::endl;
        std::cout << "-t <int>: type, 0 = test, 1 = benchmark" << std::endl;
        std::cout << "-r <rendering strategy>: singlenode/tiled/frame" << std::endl;
        std::cout << "-v <int>: vertical height of image in pixels" << std::endl;
        std::cout << "-w <int>: width of image in pixels" << std::endl;
        std::cout << "-s <int>: number of samples per pixel" << std::endl;
        std::cout << "-f <int>: number of frames to render" << std::endl;
        std::cout << "-d <int>: number of devices to use" << std::endl;
        return 0;
    }

    int image_height = find_int_arg(argc, argv, "-v", 800);
    int image_width = find_int_arg(argc, argv, "-w", 1200);
    int samples_per_pixel = find_int_arg(argc, argv, "-s", 10);

    int type = find_int_arg(argc, argv, "-t", 0);
    if (type == 0) {
        test_render(image_height, image_width, samples_per_pixel);
        return 0;
    }

    int num_frames_to_render = find_int_arg(argc, argv, "-f", 30);
    int num_devices = find_int_arg(argc, argv, "-d", 4);

    std::string rendering_strategy = find_string_option(argc, argv, "-r", std::string("singlenode"));
    if (strcmp(rendering_strategy.c_str(), "singlenode") != 0  && strcmp(rendering_strategy.c_str(), "tiled") != 0 && strcmp(rendering_strategy.c_str(), "frame") != 0) {
        std::cerr << "Unknown rendering strategy: " << rendering_strategy << std::endl;
        return 1;
    }

    benchmark_rendering(rendering_strategy, image_height, image_width, samples_per_pixel, num_frames_to_render, num_devices);
}